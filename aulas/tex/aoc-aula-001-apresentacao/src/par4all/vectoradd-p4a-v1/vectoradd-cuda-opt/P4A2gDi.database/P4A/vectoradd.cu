#include "hip/hip_runtime.h"
/* Use the Par4All accelerator run time: */
#include <p4a_accel.h>
/*
 * file for vectoradd.c
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int main(int argc, char *argv[]);
//PIPS generated variable
void p4a_launcher_main(float *a, float *b, int n);
//PIPS generated variable
void p4a_launcher_main_1(float *a, float *b, float *c, int n);
//PIPS generated variable
P4A_accel_kernel p4a_kernel_main(int i, float *a, float *b, int n), p4a_kernel_main_1(int i, float *a, float *b, float *c, int n);
//PIPS generated variable
;
//PIPS generated variable
P4A_accel_kernel_wrapper p4a_wrapper_main(int i, float *a, float *b, int n), p4a_wrapper_main_1(int i, float *a, float *b, float *c, int n);
//PIPS generated variable
;
//PIPS generated variable
;
//PIPS generated variable
;
P4A_accel_kernel_wrapper p4a_wrapper_main_1(int i, float *a, float *b, float *c, int n)
{
   // Index has been replaced by P4A_vp_0:
   i = P4A_vp_0;
   p4a_kernel_main_1(i, a, b, c, n);
}
P4A_accel_kernel p4a_kernel_main_1(int i, float *a, float *b, float *c, int n)
{
   if (i<=n-1)
      c[i] = a[i]+b[i];
}
P4A_accel_kernel_wrapper p4a_wrapper_main(int i, float *a, float *b, int n)
{
   // Index has been replaced by P4A_vp_0:
   i = P4A_vp_0;
   p4a_kernel_main(i, a, b, n);
}
P4A_accel_kernel p4a_kernel_main(int i, float *a, float *b, int n)
{
   if (i<=n-1) {
      a[i] = sinf(i)*sinf(i);
      b[i] = cosf(i)*cosf(i);
   }
}
void p4a_launcher_main_1(float *a, float *b, float *c, int n)
{
   //PIPS generated variable
   int i;
   P4A_call_accel_kernel_1d(p4a_wrapper_main_1, n, (int) i, (float *) a, (float *) b, (float *) c, (int) n);
}
void p4a_launcher_main(float *a, float *b, int n)
{
   //PIPS generated variable
   int i;
   P4A_call_accel_kernel_1d(p4a_wrapper_main, n, (int) i, (float *) a, (float *) b, (int) n);
}
int main(int argc, char *argv[])
{
   P4A_init_accel;
   
   // Size of vectors
   int n = 100000;
   
   // Input vectors
   float *a;
   float *b;
   // Output vector
   float *c;
   
   // Size, in bytes, of each vector
   size_t bytes = n*sizeof(float);
   
   // Allocate memory for each vector
   a = (float *) malloc(bytes);
   b = (float *) malloc(bytes);
   c = (float *) malloc(bytes);
   
   // Initialize content of input vectors, vector a[i] = sin(i)^2 vector b[i] = cos(i)^2
   int i;
   p4a_launcher_main(a, b, n);
   p4a_launcher_main_1(a, b, c, n);
   
   // Sum up vector c and print result divided by n, this should equal 1 within error
   float sum = 0;
   for(i = 0; i <= n-1; i += 1)
      sum += c[i];
   sum = sum/n;
   printf("final result: %f\n", sum);
   
   // Release memory
   free(a);
   free(b);
   free(c);

   return 0;
}
